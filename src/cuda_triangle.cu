#include "hip/hip_runtime.h"
// cuda_triangle.cu
//CUDA-compatible version
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

// forward declare what we need from the rasterizer
class TGAImage;
class TGAColor;
template<size_t DIM, typename T> struct vec;
typedef vec<3, float> Vec3f;
typedef vec<4, float> Vec4f;

// CUDA-compatible 3D vector
struct CudaVec3 {
    float x, y, z;
    
    __device__ __host__ CudaVec3() : x(0), y(0), z(0) {}
    __device__ __host__ CudaVec3(float x_, float y_, float z_) : x(x_), y(y_), z(z_) {}
};

//  4D vector  
struct CudaVec4 {
    float x, y, z, w;
    
    __device__ __host__ CudaVec4() : x(0), y(0), z(0), w(1) {}
    __device__ __host__ CudaVec4(float x_, float y_, float z_, float w_) : x(x_), y(y_), z(z_), w(w_) {}
};

// simple color structure
struct CudaColor {
    unsigned char r, g, b;
    
    __device__ __host__ CudaColor() : r(0), g(0), b(0) {}
    __device__ __host__ CudaColor(unsigned char r_, unsigned char g_, unsigned char b_) 
        : r(r_), g(g_), b(b_) {}
};

// GPU barycentric calculation - returns simple struct
__device__ 
CudaVec3 cuda_barycentric(float ax, float ay, float bx, float by, 
                         float cx, float cy, float px, float py) {
    float s0x = cx - ax;
    float s0y = cy - ay;
    float s1x = bx - ax;
    float s1y = by - ay;
    float s2x = ax - px;
    float s2y = ay - py;
    
    float cross_z = s0x * s1y - s0y * s1x;
    
    if (fabsf(cross_z) < 1e-6f) {
        return CudaVec3(-1, 1, 1);  // Degenerate triangle
    }
    
    float u = (s1y * s2x - s1x * s2y) / cross_z;
    float v = (s0x * s2y - s0y * s2x) / cross_z;
    
    return CudaVec3(1.0f - u - v, v, u);
}

// CUDA kernel for triangle rasterization
__global__
void triangle_raster_kernel(CudaVec4* vertices,
                           unsigned char* framebuffer,
                           float* zbuffer,
                           int width, int height,
                           CudaColor color)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    // Perspective divide
    float v0x = vertices[0].x / vertices[0].w;
    float v0y = vertices[0].y / vertices[0].w;
    float v0z = vertices[0].z;
    
    float v1x = vertices[1].x / vertices[1].w;
    float v1y = vertices[1].y / vertices[1].w;
    float v1z = vertices[1].z;
    
    float v2x = vertices[2].x / vertices[2].w;
    float v2y = vertices[2].y / vertices[2].w;
    float v2z = vertices[2].z;
    
    // Barycentric coordinates
    CudaVec3 bary = cuda_barycentric(v0x, v0y, v1x, v1y, v2x, v2y, (float)x, (float)y);
    
    if (bary.x < 0 || bary.y < 0 || bary.z < 0) return;
    
    // Depth interpolation
    float z = v0z * bary.x + v1z * bary.y + v2z * bary.z;
    float w = vertices[0].w * bary.x + vertices[1].w * bary.y + vertices[2].w * bary.z;
    float depth = z / w;
    
    int idx = y * width + x;
    
    // Simple depth test
    if (depth > zbuffer[idx]) return;
    
    zbuffer[idx] = depth;
    framebuffer[idx * 3 + 0] = color.r;
    framebuffer[idx * 3 + 1] = color.g;
    framebuffer[idx * 3 + 2] = color.b;
}

// C++ wrapper class (host-only)
class CudaTriangleRasterizer {
private:
    CudaVec4* d_vertices;
    unsigned char* d_framebuffer;
    float* d_zbuffer;
    int width, height;
    bool initialized;
    
public:
    CudaTriangleRasterizer(int w, int h) : width(w), height(h), initialized(false) {
        hipError_t err;
        
        err = hipMalloc(&d_vertices, 3 * sizeof(CudaVec4));
        if (err != hipSuccess) {
            printf("CUDA malloc vertices failed: %s\n", hipGetErrorString(err));
            return;
        }
        
        err = hipMalloc(&d_framebuffer, width * height * 3);
        if (err != hipSuccess) {
            printf("CUDA malloc framebuffer failed: %s\n", hipGetErrorString(err));
            return;
        }
        
        err = hipMalloc(&d_zbuffer, width * height * sizeof(float));
        if (err != hipSuccess) {
            printf("CUDA malloc zbuffer failed: %s\n", hipGetErrorString(err));
            return;
        }
        
        initialized = true;
        printf("CUDA rasterizer initialized: %dx%d\n", width, height);
    }
    
    ~CudaTriangleRasterizer() {
        if (initialized) {
            hipFree(d_vertices);
            hipFree(d_framebuffer);
            hipFree(d_zbuffer);
        }
    }
    
    bool isInitialized() const { return initialized; }
    
    void clear() {
        if (!initialized) return;
        
        hipMemset(d_framebuffer, 0, width * height * 3);
        
        // Initialize zbuffer to far plane
        float far_value = 1000.0f;
        float* h_init = new float[width * height];
        for (int i = 0; i < width * height; i++) {
            h_init[i] = far_value;
        }
        hipMemcpy(d_zbuffer, h_init, width * height * sizeof(float), hipMemcpyHostToDevice);
        delete[] h_init;
    }
    
    void renderTriangle(float v0x, float v0y, float v0z, float v0w,
                       float v1x, float v1y, float v1z, float v1w,
                       float v2x, float v2y, float v2z, float v2w,
                       unsigned char r, unsigned char g, unsigned char b) {
        if (!initialized) return;
        
        CudaVec4 vertices[3] = {
            CudaVec4(v0x, v0y, v0z, v0w),
            CudaVec4(v1x, v1y, v1z, v1w),
            CudaVec4(v2x, v2y, v2z, v2w)
        };
        CudaColor color(r, g, b);
        
        hipMemcpy(d_vertices, vertices, 3 * sizeof(CudaVec4), hipMemcpyHostToDevice);
        
        dim3 blockSize(16, 16);
        dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                     (height + blockSize.y - 1) / blockSize.y);
        
        triangle_raster_kernel<<<gridSize, blockSize>>>(
            d_vertices, d_framebuffer, d_zbuffer, width, height, color
        );
        
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA kernel error: %s\n", hipGetErrorString(err));
        }
    }
    
    void copyToCPU(unsigned char* host_framebuffer, float* host_zbuffer) {
        if (!initialized) return;
        
        hipMemcpy(host_framebuffer, d_framebuffer, width * height * 3, hipMemcpyDeviceToHost);
        hipMemcpy(host_zbuffer, d_zbuffer, width * height * sizeof(float), hipMemcpyDeviceToHost);
    }
    
    void synchronize() {
        if (initialized) {
            hipDeviceSynchronize();
        }
    }
};

// global instance for Engine to use
static CudaTriangleRasterizer* g_cuda_rasterizer = nullptr;

// include the rasterizer headers here so we can use the types
#include "geometry.h"
#include "tgaimage.h"

// C interface functions for Engine.cpp
extern "C" {
    bool initCudaRasterizer(int width, int height) {
        if (g_cuda_rasterizer) delete g_cuda_rasterizer;
        
        g_cuda_rasterizer = new CudaTriangleRasterizer(width, height);
        return g_cuda_rasterizer->isInitialized();
    }
    
    void cleanupCudaRasterizer() {
        if (g_cuda_rasterizer) {
            delete g_cuda_rasterizer;
            g_cuda_rasterizer = nullptr;
        }
    }
    
    void cudaClearBuffers() {
        if (g_cuda_rasterizer) {
            g_cuda_rasterizer->clear();
        }
    }
    
    void cudaRenderTriangle(const Vec4f& v0, const Vec4f& v1, const Vec4f& v2, const TGAColor& tga_color) {
        if (g_cuda_rasterizer) {
            // extract values from Vec4f and TGAColor
            TGAColor& color = const_cast<TGAColor&>(tga_color);
            
            g_cuda_rasterizer->renderTriangle(
                v0[0], v0[1], v0[2], v0[3],
                v1[0], v1[1], v1[2], v1[3], 
                v2[0], v2[1], v2[2], v2[3],
                color.bgra[2], color.bgra[1], color.bgra[0]  // BGR to RGB
            );
        }
    }
    
    void cudaCopyResults(TGAImage& framebuffer, TGAImage& zbuffer) {
        if (!g_cuda_rasterizer) return;
        
        int width = framebuffer.get_width();
        int height = framebuffer.get_height();
        
        unsigned char* h_framebuffer = new unsigned char[width * height * 3];
        float* h_zbuffer = new float[width * height];
        
        g_cuda_rasterizer->synchronize();
        g_cuda_rasterizer->copyToCPU(h_framebuffer, h_zbuffer);
        
        for (int y = 0; y < height; y++) {
            for (int x = 0; x < width; x++) {
                int idx = y * width + x;
                
                TGAColor color(h_framebuffer[idx * 3 + 0],   // R
                              h_framebuffer[idx * 3 + 1],   // G  
                              h_framebuffer[idx * 3 + 2]);  // B
                framebuffer.set(x, y, color);
                
                unsigned char depth = (unsigned char)(h_zbuffer[idx] / 10.0f);
                zbuffer.set(x, y, TGAColor(depth));
            }
        }
        
        delete[] h_framebuffer;
        delete[] h_zbuffer;
    }
}